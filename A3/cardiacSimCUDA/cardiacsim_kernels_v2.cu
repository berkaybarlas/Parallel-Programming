/* 
 * Solves the Panfilov model using an explicit numerical scheme.
 * Based on code orginally provided by Xing Cai, Simula Research Laboratory 
 * and reimplementation by Scott B. Baden, UCSD
 * 
 * Modified and  restructured by Didem Unat, Koc University
 *
 * Refer to "Detailed Numerical Analyses of the Aliev-Panfilov Model on GPGPU"
 * https://www.simula.no/publications/detailed-numerical-analyses-aliev-panfilov-model-gpgpu
 * by Xing Cai, Didem Unat and Scott Baden
 *
 */

 
#include <hip/hip_runtime.h>
#include <stdio.h>
 #include <assert.h>
 #include <stdlib.h>
 #include <iostream>
 #include <iomanip>
 #include <string.h>
 #include <math.h>
 #include <sys/time.h>
 #include <getopt.h>
 
 using namespace std;
 
 // External functions
 extern "C" void splot(double *E, double T, int niter, int m, int n);
 
 void
 cmdLine(int argc, char *argv[], double &T, int &n, int &px, int &py, int &plot_freq, int &no_comm, int &num_threads);
 
 // Utilities
 //
 // Timer
 // Make successive calls and take a difference to get the elapsed time.
 static const double kMicro = 1.0e-6;
 
 double getTime() {
     struct timeval TV;
     struct timezone TZ;
 
     const int RC = gettimeofday(&TV, &TZ);
     if (RC == -1) {
         cerr << "ERROR: Bad call to gettimeofday" << endl;
         return (-1);
     }
 
     return (((double) TV.tv_sec) + kMicro * ((double) TV.tv_usec));
 
 }  // end getTime()
 
 // Reports statistics about the computation
 // These values should not vary (except to within roundoff)
 // when we use different numbers of  processes to solve the problem
 double stats(double *E, int m, int n, double *_mx) {
     double mx = -1;
     double l2norm = 0;
     int i, j;
     for (j = 1; j <= m; j++) {
             for (i = 1; i <= n; i++) {
                     l2norm += E[j * (n+2) + i] * E[j * (n+2) + i];
                     if (E[j * (n+2) + i] > mx)
                             mx = E[j * (n+2) + i];
             }
     }
     *_mx = mx;
     l2norm /= (double) ((m) * (n));
     l2norm = sqrt(l2norm);
     return l2norm;
 }
 
 __global__ void ghosts(const int n, const int m, double *E_prev) {
     int j = threadIdx.x + 1;
 
     E_prev[j * (n+2)] = E_prev[j * (n+2) + 2];
     E_prev[j * (n+2) + (n + 1)] = E_prev[j * (n + 2) + (n - 1)];
 
     E_prev[j] = E_prev[2 * (n + 2) + j];
     E_prev[(m + 1) * (n + 2) + j] = E_prev[(m - 1) * (n + 2) + j];
 }

 __global__ void pde_ode(const double a, const double kk, const double dt, const int n, const int m, double *E, double *E_prev, double *R, 
    const double epsilon, const double M1, const double M2, const double b, const double alpha) {
    
    int i = threadIdx.x + 1;
    int j = blockIdx.x + 1;
    int index = j * (n + 2) + i;
     
    E[index] = E_prev[index] + alpha * (E_prev[index + 1] + E_prev[index - 1] - 4 * E_prev[index] + E_prev[index + m + 2] + E_prev[index - (m + 2)]);
    E[index] = E[index] - dt * (kk * E[index] * (E[index] - a) * (E[index] - 1) + E[index] * R[index]);
    R[index] = R[index] + dt * (epsilon + M1 * R[index] / (E[index] + M2)) * (-R[index] - kk * E[index] * (E[index] - b - 1));
 }
 
 void simulate(double *E, double *E_prev, double *R,
               const double alpha, const int n, const int m, const double kk,
               const double dt, const double a, const double epsilon,
               const double M1, const double M2, const double b) {
     /*
      * Copy data from boundary of the computational box
      * to the padding region, set up for differencing
      * on the boundary of the computational box
      * Using mirror boundaries
      */
 
     ghosts<<<1, n>>>(n, m, E_prev); 
     pde_ode<<<m, n>>>(a, kk, dt, n, m, E, E_prev, R, epsilon, M1, M2, b, alpha);
 }
 // Define Kernels
 
 // __global__ void 
 // __device__ 
 // Main program
 int main(int argc, char **argv) {
     /*
      *  Solution arrays
      *   E is the "Excitation" variable, a voltage
      *   R is the "Recovery" variable
      *   E_prev is the Excitation variable for the previous timestep,
      *      and is used in time integration
      */
     double *E, *R, *E_prev;
 
     // Various constants - these definitions shouldn't change
     const double a = 0.1, b = 0.1, kk = 8.0, M1 = 0.07, M2 = 0.3, epsilon = 0.01, d = 5e-5;
 
     double T = 1000.0;
     int m = 200, n = 200;
     int plot_freq = 0;
     int px = 1, py = 1;
     int no_comm = 0;
     int num_threads = 1;
 
     cmdLine(argc, argv, T, n, px, py, plot_freq, no_comm, num_threads);
     m = n;
     // Allocate contiguous memory for solution arrays
     // The computational box is defined on [1:m+1,1:n+1]
     // We pad the arrays in order to facilitate differencing on the
     // boundaries of the computation box
         E = (double *) malloc(sizeof(double) * size_t((m + 2) * (n + 2)));
     E_prev = (double *) malloc(sizeof(double) * size_t((m + 2) * (n + 2)));
     R = (double *) malloc(sizeof(double) * size_t((m + 2) * (n + 2)));
 
     int i, j;
     // Initialization
     for (j = 1; j <= m; j++)
         for (i = 1; i <= n; i++)
             E_prev[j * (n+2) + i] = R[j * (n+2) + i] = 0;
 
     for (j = 1; j <= m; j++)
         for (i = n / 2 + 1; i <= n; i++)
             E_prev[j * (n+2) + i] = 1.0;
 
     for (j = m / 2 + 1; j <= m; j++)
         for (i = 1; i <= n; i++)
             R[j * (n+2) + i] = 1.0;
 
         double dx = 1.0 / n;
 
     // For time integration, these values shouldn't change
     double rp = kk * (b + 1) * (b + 1) / 4;
     double dte = (dx * dx) / (d * 4 + ((dx * dx)) * (rp + kk));
     double dtr = 1 / (epsilon + ((M1 / M2) * rp));
     double dt = (dte < dtr) ? 0.95 * dte : 0.95 * dtr;
     double alpha = d * dt / (dx * dx);
 
     cout << "Grid Size       : " << n << endl;
     cout << "Duration of Sim : " << T << endl;
     cout << "Time step dt    : " << dt << endl;
     cout << "Process geometry: " << px << " x " << py << endl;
     if (no_comm)
         cout << "Communication   : DISABLED" << endl;
 
     cout << endl;
 
     // Start the timer
     double t0 = getTime();
 
 
     // Simulated time is different from the integer timestep number
     // Simulated time
     double t = 0.0;
     // Integer timestep number
     int niter = 0;
 
     double *d_E, *d_E_prev, *d_R;
 
     hipMalloc((void **) &d_E, sizeof(double) * (m + 2) * (n + 2));
     hipMalloc((void **) &d_E_prev, sizeof(double) * (m + 2) * (n + 2));
     hipMalloc((void **) &d_R, sizeof(double) * (m + 2) * (n + 2));
 
     hipMemcpy(d_E, E, sizeof(double) * (m + 2) * (n + 2), hipMemcpyHostToDevice);
     hipMemcpy(d_E_prev, E_prev, sizeof(double) * (m + 2) * (n + 2), hipMemcpyHostToDevice);
     hipMemcpy(d_R, R, sizeof(double) * (m + 2) * (n + 2), hipMemcpyHostToDevice);
 
     while (t < T) {
 
         t += dt;
         niter++;
 
         simulate(d_E, d_E_prev, d_R, alpha, n, m, kk, dt, a, epsilon, M1, M2, b);
 
         //swap current E with previous E
         double *tmp = d_E;
         d_E = d_E_prev;
         d_E_prev = tmp;
 
         if (plot_freq) {
             int k = (int) (t / plot_freq);
             if ((t - k * plot_freq) < dt) {
                 splot(E, t, niter, m + 2, n + 2);
             }
         }
     }//end of while loop
 
     hipMemcpy(E_prev, d_E_prev, sizeof(double) * (m + 2) * (n + 2), hipMemcpyDeviceToHost);
 
     hipFree(d_E);
     hipFree(d_E_prev);
     hipFree(d_R);  
 
     double time_elapsed = getTime() - t0;
 
     double Gflops = (double) (niter * (1E-9 * n * n) * 28.0) / time_elapsed;
     double BW = (double) (niter * 1E-9 * (n * n * sizeof(double) * 4.0)) / time_elapsed;
 
     cout << "Number of Iterations        : " << niter << endl;
     cout << "Elapsed Time (sec)          : " << time_elapsed << endl;
     cout << "Sustained Gflops Rate       : " << Gflops << endl;
     cout << "Sustained Bandwidth (GB/sec): " << BW << endl << endl;
 
     double mx;
     double l2norm = stats(E_prev, m, n, &mx);
     cout << "Max: " << mx << " L2norm: " << l2norm << endl;
 
     if (plot_freq) {
         cout << "\n\nEnter any input to close the program and the plot..." << endl;
         getchar();
     }
 
     free(E);
     free(E_prev);
     free(R);
 
     return 0;
 }
 
 void
 cmdLine(int argc, char *argv[], double &T, int &n, int &px, int &py, int &plot_freq, int &no_comm, int &num_threads) {
     /// Command line arguments
     // Default value of the domain sizes
     static struct option long_options[] = {
             {"n",          required_argument, 0, 'n'},
             {"px",         required_argument, 0, 'x'},
             {"py",         required_argument, 0, 'y'},
             {"tfinal",     required_argument, 0, 't'},
             {"plot",       required_argument, 0, 'p'},
             {"nocomm",     no_argument,       0, 'k'},
             {"numthreads", required_argument, 0, 'o'},
     };
     // Process command line arguments
     int ac;
     for (ac = 1; ac < argc; ac++) {
         int c;
         while ((c = getopt_long(argc, argv, "n:x:y:t:kp:o:", long_options, NULL)) != -1) {
             switch (c) {
 
                 // Size of the computational box
                 case 'n':
                     n = atoi(optarg);
                     break;
 
                     // X processor geometry
                 case 'x':
                     px = atoi(optarg);
 
                     // Y processor geometry
                 case 'y':
                     py = atoi(optarg);
 
                     // Length of simulation, in simulated time units
                 case 't':
                     T = atof(optarg);
                     break;
                     // Turn off communication
                 case 'k':
                     no_comm = 1;
                     break;
 
                     // Plot the excitation variable
                 case 'p':
                     plot_freq = atoi(optarg);
                     break;
 
                     // Plot the excitation variable
                 case 'o':
                     num_threads = atoi(optarg);
                     break;
 
                     // Error
                 default:
                     printf("Usage: a.out [-n <domain size>] [-t <final time >]\n\t [-p <plot frequency>]\n\t[-px <x processor geometry> [-py <y proc. geometry] [-k turn off communication] [-o <Number of OpenMP threads>]\n");
                     exit(-1);
             }
         }
     }
 }
 /* **********************************************************
  *  Author : Urvashi R.V. [04/06/2004]
  *      Modified by Didem Unat [03/23/18]
  *************************************************************/
 
 #include <stdio.h>
 
 /* Function to plot the 2D array
  * 'gnuplot' is instantiated via a pipe and
  * the values to be plotted are passed through, along
  * with gnuplot commands */
 
 FILE *gnu = NULL;
 
 void splot(double *U, double T, int niter, int m, int n) {
     int i, j;
     if (gnu == NULL) gnu = popen("gnuplot", "w");
 
     double mx = -1, mn = 32768;
     for (j = 0; j < m; j++)
         for (i = 0; i < n; i++) {
             if (U[j * m + i] > mx)
                 mx = U[j * m + i];
             if (U[j * m + i] < mn)
                 mn = U[j * m + i];
         }
 
     fprintf(gnu, "set title \"T = %f [niter = %d]\"\n", T, niter);
     fprintf(gnu, "set size square\n");
     fprintf(gnu, "set key off\n");
     fprintf(gnu, "set pm3d map\n");
     // Various color schemes
     fprintf(gnu, "set palette defined (-3 \"blue\", 0 \"white\", 1 \"red\")\n");
 
     //    fprintf(gnu,"set palette rgbformulae 22, 13, 31\n");
     //    fprintf(gnu,"set palette rgbformulae 30, 31, 32\n");
 
     fprintf(gnu, "splot [0:%d] [0:%d][%f:%f] \"-\"\n", m - 1, n - 1, mn, mx);
     for (j = 0; j < m; j++) {
         for (i = 0; i < n; i++) {
             fprintf(gnu, "%d %d %f\n", i, j, U[i * m + j]);
         }
         fprintf(gnu, "\n");
     }
     fprintf(gnu, "e\n");
     fflush(gnu);
     return;
 }
  