
#include <hip/hip_runtime.h>
#include <stdio.h>

#define RADIUS        3
#define BLOCK_SIZE    256
#define NUM_ELEMENTS  (4096*2)

__global__ void stencil_1d_simple(int *in, int *out) 
{
  // compute this thread's global index
  unsigned int i = blockDim.x * blockIdx.x + threadIdx.x + RADIUS;

  int alpha = 1; 
  int beta = 1; 

  if(i < NUM_ELEMENTS + RADIUS ){

     /* FIX ME #1 */

  }
}

__global__ void stencil_1d_improved(int *in, int *out) 
{
    __shared__ int temp[BLOCK_SIZE]; /* FIXME #2*/

    int gindex = threadIdx.x + (blockIdx.x * blockDim.x) ; /* FIXME #3*/
    int lindex = threadIdx.x ; /* FIXME #4 */

    // Read input elements into shared memory
    temp[lindex] = in[gindex];

    //Load ghost cells (halos)
    if (threadIdx.x < RADIUS) 
    {
       /* FIXME #5 */
    }

    // Make sure all threads get to this point before proceeding!
       /* FIXME #6 */	     

    // Apply the stencil
    int result = 0;
    for (int offset = -RADIUS ; offset <= RADIUS ; offset++)
        result += temp[lindex + offset];

    // Store the result
    out[gindex] = result;
}

int main()
{
  unsigned int i;
  int N = NUM_ELEMENTS + 2 * RADIUS; 
  int h_in[N], h_out[N];
  int *d_in, *d_out;

  // Initialize host data
  for( i = 0; i < (N); ++i )
    h_in[i] = 1; // With a value of 1 and RADIUS of 3, all output values should be 7

  // Allocate space on the device
  hipMalloc( &d_in,  N * sizeof(int)) ;
  hipMalloc( &d_out, N * sizeof(int)) ;

  // Copy input data to device
  hipMemcpy( d_in, h_in, N * sizeof(int), hipMemcpyHostToDevice) ;

  stencil_1d_simple<<< (NUM_ELEMENTS + BLOCK_SIZE - 1)/BLOCK_SIZE, BLOCK_SIZE >>> (d_in, d_out);
  //stencil_1d_improved<<< (NUM_ELEMENTS + BLOCK_SIZE - 1)/BLOCK_SIZE, BLOCK_SIZE >>> (d_in, d_out);

  hipMemcpy( h_out, d_out, N *  sizeof(int), hipMemcpyDeviceToHost) ;

  // Verify every out value is 7
  for( i = RADIUS; i < NUM_ELEMENTS+RADIUS; ++i )
    if (h_out[i] != RADIUS*2+1)
    {
      printf("Element h_out[%d] == %d != 7\n", i, h_out[i]);
      break;
    }

  if (i == NUM_ELEMENTS+RADIUS)
    printf("SUCCESS!\n");

  // Free out memory
  hipFree(d_in);
  hipFree(d_out);

  return 0;
}

